#include <stdio.h>
#include <hip/hip_runtime.h>


// 定义 BLOCK_NUM 和 THREAD_NUM
#define BLOCK_NUM 4  
#define THREAD_NUM 4

// __global__ void mat_mul(float *mat1, float *mat2, float *result, int M, int K, int N) {
//     const int bid = blockIdx.x;
//     const int tid = threadIdx.x;
//     const int row = bid * THREAD_NUM + tid;

//     if (row < M) {
//         for (int c = 0; c < N; c++) {
//             float sum = 0.0f;
//             for (int n = 0; n < K; n++) {
//                 sum += mat1[row * K + n] * mat2[n * N + c];
//             }
//             result[row * N + c] = sum;
//         }
//     }
// }

#define TILE_SIZE 16

// CUDA kernel for matrix multiplication using shared memory tiling
__global__ void mat_mul(const float* A, const float* B, float* C, int M, int N, int K) {
    // Shared memory for tiles of A and B
    __shared__ float shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_B[TILE_SIZE][TILE_SIZE];

    // Thread indices
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float C_value = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tiles into shared memory
        if (row < M && t * TILE_SIZE + threadIdx.x < K) {
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        } else {
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K) {
            shared_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < TILE_SIZE; ++k) {
            C_value += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result to C
    if (row < M && col < N) {
        C[row * N + col] = C_value;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <M> <K> <N>\n", argv[0]);
        return 1;
    }

    // 从命令行参数获取 M, K, N
    int M = atoi(argv[1]);
    int K = atoi(argv[2]);
    int N = atoi(argv[3]);

    printf("M: %d, K: %d, N: %d\n", M, K, N);

    float *mat1, *mat2, *result;
    float *g_mat1, *g_mat2, *g_mat_result;

    // 分配主机内存
    mat1 = (float*) malloc(M * K * sizeof(float));
    mat2 = (float*) malloc(K * N * sizeof(float));
    result = (float*) malloc(M * N * sizeof(float));

    // 初始化矩阵
    for (int i = 0; i < M * K; i++) {
        mat1[i] = (float)(rand() % 10);
    }
    for (int i = 0; i < K * N; i++) {
        mat2[i] = (float)(rand() % 10);
    }
    for (int i = 0; i < M * N; i++) {
        result[i] = 0.0f;
    }

    // 分配设备内存
    hipMalloc((void **)&g_mat1, sizeof(float) * M * K);
    hipMalloc((void **)&g_mat2, sizeof(float) * K * N);
    hipMalloc((void **)&g_mat_result, sizeof(float) * M * N);

    // 将数据从主机复制到设备
    hipMemcpy(g_mat1, mat1, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(float) * K * N, hipMemcpyHostToDevice);

    // 计算矩阵乘法
    dim3 grid((M + THREAD_NUM - 1) / THREAD_NUM); // 计算需要的 block 数量
    dim3 block(THREAD_NUM);
    mat_mul<<<grid, block>>>(g_mat1, g_mat2, g_mat_result, M, K, N);

    // 将结果从设备复制到主机
    hipMemcpy(result, g_mat_result, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result matrix (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", result[i]);
    }
    printf("\n");

    // 释放内存
    free(mat1);
    free(mat2);
    free(result);
    hipFree(g_mat1);
    hipFree(g_mat2);
    hipFree(g_mat_result);

    return 0;
}
